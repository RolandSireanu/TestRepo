
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>



__global__
void add(int n , float* x , float* y)
{
	for(int i = 0; i < n; i++)
	{
		y[i] = x[i] + y[i];
	}
}


int main()
{

	int nr = 1 << 24; // 1 million elements

	//float* a = new float[nr];
	//float* b = new float[nr];
	float *a , *b;

	hipError_t err = hipMallocManaged(&a , nr*sizeof(float));
	if(err != hipError_t::hipSuccess)
		std::cout << err <<std::endl;
	hipMallocManaged(&b , nr*sizeof(float));

	
	
	for(int i = 0; i < nr/2; i++)
	{
		a[i] = 3.0f;
		b[i] = 2.0f;
	}
	
	
	std::cout<<"A"<<std::endl;

	add<<<1 , 1>>>(nr,a,b);

	hipDeviceSynchronize();

	float maxErr = 0.0f;
	for(int i = 0; i<nr; i++)
	{
		maxErr = fmax(maxErr, fabs(b[i]-5.0f));		
	}
	
	std::cout << "Max error = " << maxErr << std::endl;

	//delete[] a;
	hipFree(a);
	//delete[] b;
	hipFree(b);


	return 0;
}
